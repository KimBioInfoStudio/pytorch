#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

constexpr int64_t BYTES_PER_THREAD = 16;
constexpr int64_t MAX_NUM_THREADS = 1024;
constexpr int64_t MIN_NUM_THREADS = 128;
constexpr int64_t WARP_SIZE = 32;

template <typename T>
__device__ inline void streamLoad128(uint4& val, const T* addr) {
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  CUDA_KERNEL_ASSERT(false);
#else
  unsigned long long int low, high;
  asm("ld.global.nc.v2.u64 {%0, %1}, [%2];"
      : "=l"(low), "=l"(high)
      : "l"(addr));
  reinterpret_cast<unsigned long long int*>(&val)[0] = low;
  reinterpret_cast<unsigned long long int*>(&val)[1] = high;
#endif
}

template <typename T>
__device__ inline void streamStore128(T* addr, const uint4& val) {
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  CUDA_KERNEL_ASSERT(false);
#else
  unsigned long long int low, high;
  low = reinterpret_cast<const unsigned long long int*>(&val)[0];
  high = reinterpret_cast<const unsigned long long int*>(&val)[1];
  asm("st.global.cs.v2.u64 [%0], {%1, %2};" : : "l"(addr), "l"(low), "l"(high));
#endif
}

static __host__ __device__ inline int64_t divUp(int64_t a, int64_t b) {
  return (a + b - 1) / b;
}

static __device__ inline bool isAligned(const void* ptr, size_t alignment) {
  uintptr_t addr = reinterpret_cast<uintptr_t>(ptr);
  return addr % alignment == 0;
}

static __global__ void fsdpAllGatherCopyOutKernel(
    void** paramPtrs,
    void* allGatherResPtr,
    int64_t totalSize,
    int64_t* blockOffsetToParamIdx,
    int64_t* blockCumSums,
    int64_t* shardDimCumSums,
    int64_t numParams,
    int64_t shardDimSum,
    int64_t blockDimSum,
    int64_t ranksPerBlock,
    int64_t worldSize) {
  const int64_t blockOffset = blockIdx.x % blockDimSum;
  const int64_t paramIdx = blockOffsetToParamIdx[blockOffset];

  for (int64_t rank = blockIdx.x / blockDimSum; rank < worldSize;
       rank += worldSize / ranksPerBlock) {
    const int64_t shardBlockCount =
        blockCumSums[paramIdx + 1] - blockCumSums[paramIdx];
    const int64_t groupSize = shardBlockCount * blockDim.x;
    const int64_t localTid =
        (blockOffset - blockCumSums[paramIdx]) * blockDim.x + threadIdx.x;

    const int64_t shardBegin = shardDimCumSums[paramIdx];
    const int64_t shardEnd = shardDimCumSums[paramIdx + 1];
    const int64_t shardLen = shardEnd - shardBegin;
    const int64_t srcOff = rank * shardDimSum + shardBegin;
    const int64_t dstOff = rank * shardLen;

    const char* srcPtr = reinterpret_cast<char*>(allGatherResPtr) + srcOff;
    char* dstPtr = &reinterpret_cast<char*>(paramPtrs[paramIdx])[dstOff];

    const int64_t alignOff =
        divUp(dstOff, BYTES_PER_THREAD) * BYTES_PER_THREAD - dstOff;
    const int64_t begin = alignOff + localTid * BYTES_PER_THREAD;
    const int64_t end =
        alignOff + (shardLen - alignOff) / BYTES_PER_THREAD * BYTES_PER_THREAD;
    const int64_t stride = groupSize * BYTES_PER_THREAD;

    for (size_t i = begin; i < end; i += stride) {
      uint4 val;
      if (isAligned(srcPtr + i, 128)) {
        streamLoad128(val, srcPtr + i);
      } else {
        for (size_t j = 0; j < BYTES_PER_THREAD; ++j) {
          reinterpret_cast<char*>(&val)[j] = srcPtr[i + j];
        }
      }
      streamStore128(&dstPtr[i], val);
    }
    if (localTid < alignOff && localTid < shardLen) {
      dstPtr[localTid] = srcPtr[localTid];
    }
    if (end + localTid < shardLen) {
      dstPtr[end + localTid] = srcPtr[end + localTid];
    }
  }
}

static int64_t geometricMean(const std::vector<int64_t>& numbers) {
  TORCH_CHECK(numbers.size() > 0);
  double logSum = 0.0;
  for (double num : numbers) {
    TORCH_CHECK(num > 0);
    logSum += log(num);
  }
  double avgLog = logSum / numbers.size();
  return exp(avgLog);
}

std::pair<at::Tensor, std::vector<int64_t*>> pack(
    std::vector<std::vector<int64_t>> vecs,
    const at::Device& device) {
  int64_t numel = 0;
  for (const auto& vec : vecs) {
    numel += vec.size();
  }

  auto packed = at::empty(
      {numel}, at::TensorOptions().dtype(at::kLong).pinned_memory(true));
  size_t offset = 0;
  for (const auto& vec : vecs) {
    memcpy(
        packed.data_ptr<int64_t>() + offset,
        vec.data(),
        sizeof(int64_t) * vec.size());
    offset += vec.size();
  }
  packed = packed.to(device, /*non_blocking=*/true);

  std::vector<int64_t*> ptrs;
  offset = 0;
  for (const auto& vec : vecs) {
    ptrs.push_back(packed.data_ptr<int64_t>() + offset);
    offset += vec.size();
  }
  return std::make_pair(packed, ptrs);
}

void fsdpAllGatherCopyOut(
    std::vector<at::Tensor> params,
    at::Tensor allGatherRes,
    int64_t worldSize) {
  const auto device = allGatherRes.device();
  const auto totalSize = allGatherRes.numel() * allGatherRes.element_size();

  TORCH_CHECK(allGatherRes.is_cuda());
  TORCH_CHECK(allGatherRes.is_non_overlapping_and_dense());

  std::vector<int64_t> paramPtrs;
  std::vector<int64_t> shardDims; // In bytes
  std::vector<int64_t> dimCumSums{0}; // In bytes
  for (size_t i = 0; i < params.size(); ++i) {
    const auto& param = params[i];
    TORCH_CHECK(param.is_non_overlapping_and_dense());
    TORCH_CHECK(param.device() == device);
    TORCH_CHECK(param.numel() > 0);
    // All params are expected to be aligned at worldSize.
    TORCH_CHECK(param.numel() % worldSize == 0);
    const auto shardDim = param.numel() * param.element_size() / worldSize;
    paramPtrs.push_back(reinterpret_cast<int64_t>(param.data_ptr()));
    shardDims.push_back(shardDim);
    dimCumSums.push_back(dimCumSums[i] + shardDim);
  }

  TORCH_CHECK(
      dimCumSums.back() * worldSize == totalSize,
      "The total byte size must be identical between params and allGatherRes");

  // To balance the throughput larger shards and waste on smaller shards, we
  // use the geometric mean of the shard dims to determine the block size.
  int64_t meanShardDim = geometricMean(shardDims);
  int64_t blockSize = divUp(meanShardDim, BYTES_PER_THREAD);
  blockSize = divUp(blockSize, WARP_SIZE) * WARP_SIZE;
  blockSize = std::min(std::max(blockSize, MIN_NUM_THREADS), MAX_NUM_THREADS);

  // TODO: this is only for A100
  constexpr int64_t maxActiveBlocks = 32 * 108;
  constexpr double smOverSubFactor = 1.75;

  // Roughly estimate the amount of blocks needed for each rank, and calculate
  // an iter factor to regularize SM over-subscription.
  int64_t iterFactor = 1;
  while (divUp(totalSize, blockSize * BYTES_PER_THREAD * iterFactor) >
         (maxActiveBlocks * smOverSubFactor)) {
    iterFactor += 1;
  }

  std::vector<int64_t> blockOffsetToParamIdx;
  std::vector<int64_t> blockCumSums{0};
  for (int64_t paramIdx = 0; paramIdx < static_cast<int64_t>(params.size());
       ++paramIdx) {
    int64_t numBlocks =
        divUp(shardDims[paramIdx], blockSize * BYTES_PER_THREAD * iterFactor);
    blockOffsetToParamIdx.insert(
        blockOffsetToParamIdx.end(), numBlocks, paramIdx);
    blockCumSums.push_back(blockCumSums.back() + numBlocks);
  }
  const auto numBlocks = blockCumSums.back();

  auto packed = pack(
      {paramPtrs, blockOffsetToParamIdx, blockCumSums, dimCumSums}, device);

  int64_t ranksPerBlock = 1;
  while (numBlocks * (worldSize / ranksPerBlock) >
             maxActiveBlocks * smOverSubFactor &&
         ranksPerBlock < worldSize) {
    ++ranksPerBlock;
  }

  dim3 blocks(numBlocks * (worldSize / ranksPerBlock), 1, 1);
  dim3 threads(blockSize, 1, 1);

  LOG(INFO) << "meanShardDim: " << meanShardDim
            << ", iterFactor: " << iterFactor
            << ", ranksPerBlock: " << ranksPerBlock << ", blocks: " << blocks.x
            << ", threads: " << threads.x;

  fsdpAllGatherCopyOutKernel<<<
      blocks,
      threads,
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      reinterpret_cast<void**>(packed.second[0]),
      allGatherRes.data_ptr(),
      totalSize,
      /*blockOffsetToParamIdx=*/packed.second[1],
      /*blockCumSums=*/packed.second[2],
      /*shardDimCumSums=*/packed.second[3],
      params.size(),
      dimCumSums.back(),
      blockCumSums.back(),
      ranksPerBlock,
      worldSize);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}
